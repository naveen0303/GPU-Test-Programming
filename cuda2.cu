#include "hip/hip_runtime.h"
//Author: Naveen Milind Chalawadi
// Input Arrays - (CPU)Arrayin		(GPU)GpAin
// Output Array - (CPU)Arrayout		(GPU)GpAout
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>


using namespace std;

//Macro for checking errors when using Cuda APIs and printing the error report on screen
#define Handle_Error(err) (HANDLE_ERROR(err))
static void Handle_Error(hipError_t err) {
	if (err != hipSuccess) {
		cout << "Error!!!!" << endl;
		cout << hipGetErrorString(err) << endl;
	}
}

//Function Definition and Declaration
__global__ void reduction_sum(float*,float*, int);
__global__ void reduction_sum(float* GpAout, float* GpAin, int m)
{
	extern __shared__ int shareddata[];
	
	//compute the global_id and local_id for each thread
	int global_id = blockIdx.x * blockDim.x + threadIdx.x;
	int local_id = threadIdx.x;
	
	// each thread loads one element from global memory to shared memory
	shareddata[local_id] = GpAin[global_id];
	__syncthreads();
	
	// do reduction in shared mem
	for (unsigned int k = blockDim.x / 2; k > 0; k >>= 1) {
		if (local_id < k) {
			shareddata[local_id] += shareddata[local_id + k];
		}
		__syncthreads();
	}
		
	// write result for this block to global memory
	if (local_id == 0) GpAout[blockIdx.x] = shareddata[0];
}

int main(int argc, char** argv)
{
	if (argc != 2)
	{
		printf("Error!!! Need an argument - Array Size\n");
		exit(1);
	}
	int m = atoi(argv[1]);
	int datasize1 = m * sizeof(float);
	float* Arrayin = (float*)malloc(datasize1);
	
	//Initialize Input Arrays (I'm just using initial value of 1 assigned to all elements in both the arrays
	for (int i = 0; i < m; i++) {
		Arrayin[i] = 1;
	}
	
	int datasize2 = 1 * sizeof(float);
	float* Arrayout = (float*)malloc(datasize2);
	Arrayout[0] = 0;

	//GPU Implementation starts here
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));	// getting properties of cuda device to get the calculate threads

	dim3 threads(prop.maxThreadsPerBlock);//calculate the gridsize and blocksizze
	dim3 blocks(m / threads.x + 1);
	int sharedBytes = threads.x * sizeof(float);

	float* GpAin, *GpAout;						//pointer to device memory
	HANDLE_ERROR(hipMalloc(&GpAin, datasize1));		// allocate memory to device pointers
	HANDLE_ERROR(hipMalloc(&GpAout, datasize2));


	HANDLE_ERROR(hipMemcpy(GpAin, Arrayin, datasize1, hipMemcpyHostToDevice));	//copy the values from CPU to GPU
	
	//Assigning clock timers to time the implementations
	hipEvent_t startTime, stopTime;
	float elapsedTime = 0;
	hipEventCreate(&startTime);
	hipEventCreate(&stopTime);

	//Perform the convolution 
	hipEventRecord(startTime);
	reduction_sum <<<blocks, threads, sharedBytes >>> (GpAout, GpAin, m);
	hipDeviceSynchronize();
	hipEventRecord(stopTime);

	//Get the elapsed time 
	hipEventSynchronize(stopTime);
	hipEventElapsedTime(&elapsedTime, startTime, stopTime);

	HANDLE_ERROR(hipMemcpy(Arrayout, GpAout, datasize1, hipMemcpyDeviceToHost));	//copy the calculated values back into the CPU from GPU

	
	hipFree(GpAin);//Free the allocated memory
	hipFree(GpAout);
	cout << Arrayout[0] << endl;
	free(Arrayin);						//free the allocated memory for input
	free(Arrayout);

	printf("\nComputation Time: %f ms", elapsedTime);
	//print the time taken for convolution using GPU

	return 0;
}
